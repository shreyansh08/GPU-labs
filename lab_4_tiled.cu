#include<stdio.h>
#include<stdlib.h>
#include<stdbool.h>
#include<string.h>
#include<hip/hip_runtime.h>
#include<time.h>
#include<hip/hip_runtime_api.h>

#include <hip/device_functions.h>

#define MASK_WIDTH 3    //Here MASK_WIDTH = MASK_HEIGHT = 2*N + 1 where N is half-width of the chosen square mask
#define TILE_WIDTH 8   //Tile Width = Tile Height
#define W (TILE_WIDTH + MASK_WIDTH - 1) //mask_width-1 for the halo elements and tile_width for internal elements

__global__ void convolution_kernel(unsigned char *input_img, unsigned char *output_img, int height, int width)
{

    __shared__ unsigned char input_shared[W][W]; //Shared Memory required for a tile and its halo elements(3 channels)
    
    int chan;
    for(chan=0;chan<3;chan++)    //3 Channel Image
    {
        int tx = threadIdx.x;
        int ty = threadIdx.y;

        int output_row = blockIdx.x*TILE_WIDTH + tx;
        int output_col = blockIdx.y*TILE_WIDTH + ty;
        
        int input_row = output_row - MASK_WIDTH/2;
        int input_col = output_col - MASK_WIDTH/2;

        if((input_row >= 0) && (input_row < height) && (input_col >= 0) && (input_col < width))
        {
            input_shared[tx][ty] = input_img[(input_row*width + input_col)*3 + chan];  
        }
        else
        {
            if(input_row<0 && input_col<0)
	    {
		input_shared[tx][ty] = input_img[chan];
	    }
	    else if(input_row<0 && input_col<width)
	    {
                input_shared[tx][ty] = input_img[3*input_col + chan];
	    }	
	    else if(input_row<0)
	    {
		input_shared[tx][ty] = input_img[3*(width-1) + chan];
	    }
	    else if(input_row<height && input_col<0)
	    {
		input_shared[tx][ty] = input_img[input_row*width*3 + chan];
	    }
	    else if(input_row<height && input_col>width)
            {
		input_shared[tx][ty] = input_img[(input_row*width +width-1)*3 + chan];
	    }
	    else if(input_row>height && input_col<0)
	    {
		input_shared[tx][ty] = input_img[width*(height-1)*3 + chan];
	    }
	    else if(input_row>height && input_col<width)
            {
		input_shared[tx][ty] = input_img[(width*(height-1)+input_col)*3 + chan];
	    }
	    else
	    {
		input_shared[tx][ty] = input_img[(width*(height-1) + (width-1))*3 + chan];
            }
        }

	 __syncthreads();

        int i;
        if(tx<TILE_WIDTH && ty<TILE_WIDTH)
        {
            int j;
            int freq[256];
        
            for(i=0;i<256;i++)freq[i]=0;
    
            for(i=0;i<MASK_WIDTH;i++)
            {
                for(j=0;j<MASK_WIDTH;j++)
                {
                    freq[input_shared[tx+i][ty+j]]++;
                }
            }
            j=0;
            for(i=0;i<256;i++)
            {
                j=j+freq[i];
                if(j>((MASK_WIDTH*MASK_WIDTH)/2))break;    
            }
        }
  
        if(output_row<height && output_col<width)
        {
        	output_img[(output_row*width + output_col)*3 + chan] = i;
        }
        __syncthreads();
    }
    
}
unsigned char* readImg(char *filename,int *height_out, int *width_out, int *maxval_out)
{
    FILE *fptr;
    char buf[16];
    
    int i,j,k;
    int height,width,maxval;
    unsigned char *input_img;

    int index = 0;
    fptr =  fopen(filename,"rb");
    
    if(!fptr)
    {
        printf("Unable to open file '%s'\n",filename);
        return NULL;
    }
   
    if (!fgets(buf, sizeof(buf), fptr))
    {
        printf("Error reading format\n");
        return NULL;
    }

    int c = getc(fptr);
    while(c == '#'){
    while(getc(fptr)!='\n');
         c = getc(fptr);
    }
    ungetc(c, fptr);
	
    if(fscanf(fptr,"%d %d",&height,&width) !=2){
	printf("ERROR Reading Dimension\n");
	return NULL;
    }

    if(fscanf(fptr,"%d",&maxval)!=1){
	printf("ERROR Reading MAXDEPTH\n");
	return NULL;
    }
    
    while(fgetc(fptr) != '\n');
           
    printf("%d\t%d\t%d\n",height,width,maxval);
    
    int pix = width*height;

    input_img = (unsigned char*)(malloc((3*pix)*sizeof(unsigned char)));

    
    if (fread(input_img,sizeof(unsigned char),3*pix, fptr) != 3*pix)
    {
         printf("Error loading image '%s'\n", filename);
         return NULL;
    }

    *height_out = height;
    *width_out = width;
    *maxval_out = maxval;

    fclose(fptr);
    printf("Image read successfully\n");
    return input_img;
}

int writeImg(int width, int height, int maxval, unsigned char *output_img)
{
    FILE *fptr;
    fptr = fopen("output.ppm","wb");
    if(!fptr)
    {
        printf("Error opening file\n");
        return 0;
    }
    fprintf(fptr,"P6\n");

    fprintf(fptr, "%d %d\n",height,width);

    // rgb component depth
    fprintf(fptr, "%d\n",maxval);

    int pix = 3*width*height;
    // pixel data
    int j;
    if((j=fwrite(output_img,sizeof(unsigned char),pix, fptr))!=pix)
    {
	printf("ERROR WRITING %d\n",j);
    }
    fclose(fptr);
    return 1;
}

int main(int argc, char **argv)
{
    int height,width,maxval;
    unsigned char *input_img,*output_img;
    unsigned char *dev_input_img,*dev_output_img;

    input_img = readImg(argv[1], &height, &width, &maxval);

    //printf("%d %d %d\n",height,width,maxval);
    
    int pix = width*height;

    hipMalloc((void**)&dev_input_img,3*pix*sizeof(unsigned char));
    hipMalloc((void**)&dev_output_img,3*pix*sizeof(unsigned char));

    //if(input_img == NULL)printf("WHY\n");

    hipMemcpy(dev_input_img,input_img,3*pix*sizeof(unsigned char),hipMemcpyHostToDevice);

    int BlockX = ceil(((float)height)/TILE_WIDTH);
    int BlockY = ceil(((float)width)/TILE_WIDTH);

    dim3 dimGrid(BlockX, BlockY,1);
    dim3 dimBlock(W,W,1);


    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);	
    
    hipEventRecord(start);
    convolution_kernel<<<dimGrid,dimBlock>>>(dev_input_img, dev_output_img, height, width);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    float parallel_time = 0;
    hipEventElapsedTime(&parallel_time, start, stop);
    
    hipDeviceSynchronize();

    output_img = (unsigned char*)(malloc(3*pix*sizeof(unsigned char)));

    hipMemcpy(output_img,dev_output_img,3*pix*sizeof(unsigned char),hipMemcpyDeviceToHost);

    int i = writeImg(width,height,maxval,output_img);
    if(i==0)
	return 1;
    hipError_t error = hipGetLastError();
    if(error != hipSuccess)
    {
      // print the CUDA error message and exit
      printf("CUDA error: %s\n", hipGetErrorString(error));
      exit(-1);
    }
    printf("%f\n",parallel_time);
}