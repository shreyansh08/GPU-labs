#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <time.h>

__global__ void add(int N, double *a,double *b)
{
    int tid = blockIdx.x*blockDim.x + threadIdx.x;
    if(tid < N)
    {
        b[tid] = a[tid]*a[tid];
    }

}

int main(int argc, char *argv[])
{
    int N;  //Problem Size
    int T = 10, B = 1;            // threads per block/blocks per grid
    double *a,*b;
    double *dev_a, *dev_b;

    for(N=10000000;N<=100000000;N=N+10000000)
    {
        printf("N = %d\n",N);
        a = (double*)malloc(sizeof(double)*N);
        b = (double*)malloc(sizeof(double)*N);

        hipMalloc((void**)&dev_a,N * sizeof(double));
        hipMalloc((void**)&dev_b,N * sizeof(double));

        for(int i=0;i<N;i++)
        {
                // load arrays with some numbers
                a[i] = i;
        }

        hipMemcpy(dev_a, a , N*sizeof(double),hipMemcpyHostToDevice);

        T = 1024;
        B = ceil(double(N)/T);

        clock_t start_time = clock();
        add<<<B,T>>>(N,dev_a,dev_b);
        hipDeviceSynchronize();
        clock_t end_time = clock();
        
        double parallel_time = (double(end_time-start_time)/CLOCKS_PER_SEC);
        hipMemcpy(b,dev_b,N*sizeof(double),hipMemcpyDeviceToHost);

        start_time = clock();
        int i;
        for(i=0;i<N;i++)
        {
            b[i] = a[i] * a[i];
        }
        end_time = clock();

        double serial_time  = (double(end_time-start_time)/CLOCKS_PER_SEC);

        double speedup = serial_time/parallel_time;
        printf("N=%d, parallel_time = %lf, serial_time = %lf, speedup = %lf\n",N,parallel_time,serial_time,speedup);
        //printf("%lf %lf\n",a[5],b[5]);
        free(a);
        free(b);

        hipFree(dev_a); // clean up
        hipFree(dev_b);
    }

        return 0;
}